#include <cstdlib>
#include <iostream>
#include <cmath>
#include <queue>

#include "phil_math.h"
#include "utilities.h"

void find_neighbors(int point_A, int* vectors, int N, float epsilon, int* output_cluster_IDs, std::queue<int> &neighbors) {
    for (int point_B = 0; point_B < N; point_B++) {
            if (point_A == point_B) continue;
            if (output_cluster_IDs[point_B] != -2) continue;   // previously processed
            if (euclidean_distance_3D(point_A, point_B, vectors) < epsilon) {   // same cluster
                neighbors.push(point_B);
            }
        }
    return;
}

void dbscan_serial(int min_neighbors, float epsilon, int* vectors, int vector_length, int N,
                    int* output_cluster_IDs) {
// Refer to README.md

    std::cout << "Running DBSCAN\tepsilon: " << epsilon << "\tmin_neighbors: " << min_neighbors << "\n\n";

    // Noise will be ID'ed as -1.
    // Clusters will be 0+
    int current_cluster_ID = 0;
    for (int point_A = 0; point_A < N; point_A++) {

        if (output_cluster_IDs[point_A] != -2) continue;    // previously processed

        std::queue<int> neighbors;

        find_neighbors(point_A, vectors, N, epsilon, output_cluster_IDs, neighbors);        // updates neighbors queue
        std::cout << "Point " << point_A << " Neighbor list:\n";
        showq(neighbors);
        std::cout << '\n';

        std::cout << "neighbors.size(): " << neighbors.size() << "\n";
        if (neighbors.size() < min_neighbors) {                         // label as noise
            output_cluster_IDs[point_A] = -1;
            continue;
        }

        // Process neighbors
        output_cluster_IDs[point_A] = current_cluster_ID;
        while (!neighbors.empty()) {
            int neighbor = neighbors.front();
            neighbors.pop();

            std::cout << "\ncurrent neighbor: " << neighbor << " cluster ID: " << output_cluster_IDs[neighbor] << "\n"; 
            if (output_cluster_IDs[neighbor] != -2) continue;   // previously processed
            
            output_cluster_IDs[neighbor] = current_cluster_ID; // label as this cluster

            // Queue neighbors of neighbors
            std::queue<int> new_neighbors;
            find_neighbors(neighbor, vectors, N, epsilon, output_cluster_IDs, new_neighbors);   // updates neighbors queue
            if (new_neighbors.size() >= min_neighbors) {                                        // `neighbor` is a core point,
                while (!new_neighbors.empty()) {                                                // so we can expand its neighbors
                    neighbors.push(new_neighbors.front());
                    new_neighbors.pop();
                }
            }
        }

        current_cluster_ID += 1;

        std::cout << "Cluster IDs:\n";
        show_numbered(output_cluster_IDs, N);
    }

}

int main() {

    // `N` vectors with `vector_length` components
    int N = 32;
    int vector_length = 3;
    int vectors[N * vector_length];

    // DBSCAN parameters
    int min_neighbors = 3;
    float epsilon = 30;
    
    // Populate them with random integers within [0, 99], seed for reproducibility
    int upper_bound = 100;
    srand(1);                       // set seed
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < vector_length; j++) {
            vectors[i * 3 + j] = rand() % upper_bound;
        }
    }

    // Print vectors
    std::cout << "Vectors:\n";
    show(vectors, vector_length, N);

    // Points will be labeled with the cluster ID (N points, first value is point ID, second is cluster ID)
    // undefined    -> -2
    // noise        -> -1
    // cluster      ->  0+
    int output_cluster_IDs[N];
    std::fill_n(output_cluster_IDs, N, -2); // fill array with -2

    std::cout << "Cluster IDs:\n";
    show_numbered(output_cluster_IDs, N);

    dbscan_serial(min_neighbors, epsilon, vectors, vector_length, N, output_cluster_IDs);

    std::cout << "Cluster IDs:\n";
    show_numbered(output_cluster_IDs, N);

    //add<<<32, 1024>>>(vector_a, vector_b, vector_c, length);
    //hipDeviceSynchronize();
    return 0;
}
